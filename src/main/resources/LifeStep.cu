
#include <hip/hip_runtime.h>
extern "C"
__global__ void lifeStep(char** lifeData, int width, int height) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int right = (x + 1) % width;
    int left = (x + width - 1) % width;

    int top = (y + height - 1) % height;
    int down = (y + 1) % height;

    // Count alive cells.
    int aliveCells =
        lifeData[left][top] +  lifeData[x][top]  + lifeData[right][top] +
        lifeData[left][y]                        + lifeData[right][y] +
        lifeData[left][down] + lifeData[x][down] + lifeData[right][down];

    lifeData[x][y] = aliveCells == 3 || (aliveCells == 2 && lifeData[x][y]) ? 1 : 0;
}